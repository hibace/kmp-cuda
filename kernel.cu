#pragma region includes

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <hip/device_functions.h>

#pragma endregion

#pragma region defines

#define BLOCK_NUM 1
#define THREADS_NUM 512
#define MAX_THREADS_PER_BLOCK 1024

#pragma endregion

// preprocess
void getNext(char *pattern, int pattern_len, int *next)
{
	int len = 0;  // Record the length of the previous [longest matching prefix and suffix]
	int i;
	next[0] = 0; // next[0] Must be 0
	i = 1;
	// the loop calculates next[i] for i = 1 to pattern_len-1
	while (i < pattern_len)
	{
		if (pattern[i] == pattern[len])
		{
			len++;
			next[i] = len;
			i++;
		}
		else // (pat[i] != pat[len])
		{
			if (len == 0)
			{
				next[i] = len; // No match
				i++;
			}
			else // in case (len != 0)
			{
				len = next[len - 1];
			}
		}
	}
}

// kmp algorithm https://habr.com/ru/post/307220/
__device__ void KMP(char *pattern, int pattern_len, char *array, int array_len, int *answer, int *next, int cursor, int end)
{
	//  Each thread processes a pattern_len number, ie the step size of index is id*pattern_len
	int j = 0;//j as index for pattern
	//cursor as index for array
	while (cursor < end)
	{
		if (pattern[j] == array[cursor])
		{
			j++;
			cursor++;
		}
		if (j == pattern_len)
		{
			//printf("Found pattern at index %d \n", i - j);
			answer[cursor - j] = 1;
			j = next[j - 1];
		}
		// mismatch after j matches
		else if (pattern[j] != array[cursor])
		{
			// Do not match next[0..next[j-1]] characters,
			// they will match anyway
			if (j != 0)
				j = next[j - 1];
			else
				cursor = cursor + 1;
		}
	}
}

__global__ void kmp_kernel(char *arrayIn, char *patternIn, int *answerIn, int *next, int array_len, int pattern_len)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	int offset = 2 * pattern_len;
	int cursor, end;

	if (id < 0.5*(array_len / pattern_len))
	{
		cursor = id * offset;
		end = id * offset + offset;
	}
	else
	{ //aid thread
		cursor = (id % ((array_len / pattern_len) / 2))*offset + offset - pattern_len;
		end = (id % ((array_len / pattern_len) / 2))*offset + offset + pattern_len;
	}

	KMP(patternIn, pattern_len, arrayIn, array_len, answerIn, next, cursor, end);
	//__shared__ char array[blockDim.x+2*pattern_len];
}

int main()
{
	//error handling
	hipError_t r;
	//host copies declaration
	char *array, *pattern; int *answer;
	//device copies declaration
	char *d_array, *d_pattern; int *d_answer;

	//input file operations & host arrays
	FILE * infile = fopen("logs.txt", "r");
	if (infile == NULL) {
		printf("ERROR:Could not open file '%s'.\n", "infile");
		exit(-1);
	}

	FILE * patternFile = fopen("pattern.txt", "r");
	if (patternFile == NULL) {
		printf("ERROR:Could not open file '%s'.\n", "patternFile");
		exit(-1);
	}
	char readTemp, patternReadTemp;
	int array_len = 0; int pattern_len = 0;
	while ((readTemp = fgetc(infile)) != EOF) array_len++;
	while ((patternReadTemp = fgetc(patternFile)) != EOF) pattern_len++;
	
	if (pattern_len > array_len || pattern_len < 0 || array_len < 0) { printf("ERROR INPUT!"); return 0; }
	bool zero_flag = false;
	if (pattern_len == 0 && array_len == 0) zero_flag = true;

	
	fseek(infile, 0, SEEK_SET);

	array = (char*)malloc(array_len * sizeof(char));
	pattern = (char*)malloc(pattern_len * sizeof(char));
	answer = (int*)malloc(array_len * sizeof(int));
	int readTemp1 = 0;
	while ((readTemp = fgetc(infile)) != EOF) { array[readTemp1] = readTemp; readTemp1++; }
	fclose(infile);

	fseek(patternFile, 0, SEEK_SET);
	int readTemp2 = 0;
	while ((patternReadTemp = fgetc(patternFile)) != EOF) { pattern[readTemp2] = patternReadTemp; readTemp2++; }
	fclose(patternFile);
	for (readTemp1 = 0; readTemp1 < array_len; readTemp1++) answer[readTemp1] = 0;

	//device arrays allocation
	r = hipMalloc((void**)&d_array, sizeof(char)*array_len);
	printf("hipMalloc d_array : %s\n", hipGetErrorString(r));
	r = hipMalloc((void**)&d_pattern, sizeof(char)*pattern_len);
	printf("hipMalloc d_pattern : %s\n", hipGetErrorString(r));
	r = hipMalloc((void**)&d_answer, sizeof(int)*array_len);
	printf("hipMalloc d_answer : %s\n", hipGetErrorString(r));

	int* r_next = (int*)malloc(pattern_len * sizeof(int));
	//device
	int* next;
	r = hipMalloc((void**)&next, sizeof(int)*pattern_len);
	printf("hipMalloc next : %s\n", hipGetErrorString(r));
	//preprocessing
	getNext(pattern, pattern_len, r_next);

	//cudaMemcpy for parameters
	r = hipMemcpy(d_array, array, sizeof(char)*array_len, hipMemcpyHostToDevice);
	printf("Memory copy H->D d_array : %s\n", hipGetErrorString(r));
	r = hipMemcpy(d_pattern, pattern, sizeof(char)*pattern_len, hipMemcpyHostToDevice);
	printf("Memory copy H->D d_pattern : %s\n", hipGetErrorString(r));
	//copy for next
	r = hipMemcpy(next, r_next, sizeof(int)*pattern_len, hipMemcpyHostToDevice);
	printf("Memory copy H->D d_pattern : %s\n", hipGetErrorString(r));

	//=========================================================================//
		//Each thread processes a string of pattern length
	int threads = (array_len / pattern_len) <= MAX_THREADS_PER_BLOCK ? (array_len / pattern_len) : MAX_THREADS_PER_BLOCK;
	
	int blocks = (threads / 1024) + 1;
	//call kernel
	kmp_kernel <<< blocks, threads >>> (d_array, d_pattern, d_answer, next, array_len, pattern_len);

	r = hipDeviceSynchronize();
	printf("Device synchronize : %s\n", hipGetErrorString(r));

	//cudaMemcpy for result
	r = hipMemcpy(answer, d_answer, sizeof(int)*array_len, hipMemcpyDeviceToHost);
	printf("Memory copy D->H answer : %s\n", hipGetErrorString(r));

	//test
	//int test;
	//for (test = 0; test < array_len; test++) printf("pos[%d]=%d\n", test, answer[test]);

	//output file operations
	FILE * outfile = fopen("output.txt", "w+");
	if (outfile == NULL) {
		printf("ERROR:Could not open file '%s'.\n", "outfile");
		exit(-1);
	}
	if (zero_flag == false)
	{
		int writeTemp;
		bool flag = 0;
		for (writeTemp = 0; writeTemp < array_len; writeTemp++)
			if (answer[writeTemp] == 1)
			{
				if (flag == 0) flag = 1;
				fprintf(outfile, "Found at position %d\n", writeTemp);
			}
		if (flag == 0) fprintf(outfile, "Not found.");
	}
	else fprintf(outfile, "Null input.");
	fclose(outfile);

	//pointers free (host&device)
	free(array); free(pattern); free(answer); free(r_next);
	hipFree(d_array); hipFree(d_pattern); hipFree(d_answer); hipFree(next);

	return 1;
}
